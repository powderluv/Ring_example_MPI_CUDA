#include "hip/hip_runtime.h"
#include <chrono>
#include <stdio.h>
#include <cassert>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "host_to_device.hpp"

typedef std::chrono::high_resolution_clock Clock;

__global__
void __compute_in_kernel__(int r, int c, float* d_array, int world_rank)
{
  // do some computation on the device
  for(int i = 0; i<r*c; i++)
  {
	d_array[i] += 1;
	assert((float)(world_rank+1+i)==d_array[i]);
  }  
}

void compute(int r, int c, float** array, int world_rank)
{
  float *d_array;

  hipMalloc((void **)&d_array, r*c*sizeof(float));

  auto start_h2d = Clock::now();

  hipMemcpy(d_array, &(array[0][0]), r*c*sizeof(float), hipMemcpyHostToDevice);

  auto end_h2d = Clock::now();
  auto time_h2d = std::chrono::duration_cast<std::chrono::duration<double>>(end_h2d - start_h2d).count();
  std::cout << "rank " << world_rank << ": time spent on H2D mem copy is :"<< time_h2d << " seconds \n";

  __compute_in_kernel__<<<1,1>>>(r, c, d_array, world_rank);

  std::cout << "array is computed at rank [" << world_rank << "]'s device.\n";

  auto start_d2h = Clock::now();

  hipMemcpy(&(array[0][0]), d_array, r*c*sizeof(float), hipMemcpyDeviceToHost);

  auto end_d2h = Clock::now();
  auto time_d2h = std::chrono::duration_cast<std::chrono::duration<double>>(end_d2h - start_d2h).count();
  std::cout << "rank " << world_rank << ": time spent on d2h mem copy is :"<< time_d2h << " seconds \n";
}

