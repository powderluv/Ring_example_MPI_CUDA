#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cassert>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "host_to_device.hpp"

__global__
void __compute_in_kernel__(int r, int c, float* d_array, int world_rank)
{
  // do some computation on the device
  for(int i = 0; i<r*c; i++)
  {
	d_array[i] += 1;
	assert((float)(world_rank+1+i)==d_array[i]);
  }  
}

void compute(int r, int c, float** array, int world_rank)
{
  float *d_array;

  hipMalloc((void **)&d_array, r*c*sizeof(float));
  hipMemcpy(d_array, &(array[0][0]), r*c*sizeof(float), hipMemcpyHostToDevice);
  
  __compute_in_kernel__<<<1,1>>>(r, c, d_array, world_rank);

  std::cout << "array is computed at rank [" << world_rank << "]'s device.\n";

  hipMemcpy(&(array[0][0]), d_array, r*c*sizeof(float), hipMemcpyDeviceToHost);

}

